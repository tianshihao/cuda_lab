#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdlib>
#include <iostream>

#include "timing.h"

namespace cuda_lab::timing {

void CheckCudaError(hipError_t const err, char const* msg) {
  if (err != hipSuccess) {
    std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

__global__ void VectorAddKernel(int const* d_a, int const* d_b, int* d_c,
                                unsigned int N) {
  auto idx{blockIdx.x * blockDim.x + threadIdx.x};
  if (idx < N) {
    d_c[idx] = d_a[idx] + d_b[idx];
  }
}

void VectorAdd(int const* h_a, int const* h_b, int* h_c, unsigned int N) {
  int *d_a, *d_b, *d_c;

  // Assign device memory
  hipError_t err{hipMalloc((void**)&d_a, N * sizeof(int))};
  CheckCudaError(err, "Failed to allocate device memory for d_a");

  err = hipMalloc((void**)&d_b, N * sizeof(int));
  CheckCudaError(err, "Failed to allocate device memory for d_b");

  err = hipMalloc((void**)&d_c, N * sizeof(int));
  CheckCudaError(err, "Failed to allocate device memory for d_c");

  // Transfer data from host to device
  err = hipMemcpy(d_a, h_a, N * sizeof(int),
                   hipMemcpyKind::hipMemcpyHostToDevice);
  CheckCudaError(err, "Failed to copy h_a to d_a");

  err = hipMemcpy(d_b, h_b, N * sizeof(int),
                   hipMemcpyKind::hipMemcpyHostToDevice);
  CheckCudaError(err, "Failed to copy h_b to d_b");

  // Define the block size and grid size
  int blockSize{256};
  int gridSize{static_cast<int>((N + blockSize - 1) / blockSize)};

  // Calling the kernel
  VectorAddKernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);

  // Check for kernel errors
  err = hipGetLastError();
  CheckCudaError(err, "Kernel execution failed");

  // Transfer data from device to host
  err = hipMemcpy(h_c, d_c, N * sizeof(int),
                   hipMemcpyKind::hipMemcpyDeviceToHost);
  CheckCudaError(err, "Failed to copy d_c to h_c");

  // Free device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

void VectorAdd(hipStream_t stream, int const* h_a, int const* h_b, int* h_c,
               unsigned int N) {
  int *d_a, *d_b, *d_c;

  // Assign device memory
  hipError_t err{hipMalloc((void**)&d_a, N * sizeof(int))};
  CheckCudaError(err, "Failed to allocate device memory for d_a");

  err = hipMalloc((void**)&d_b, N * sizeof(int));
  CheckCudaError(err, "Failed to allocate device memory for d_b");

  err = hipMalloc((void**)&d_c, N * sizeof(int));
  CheckCudaError(err, "Failed to allocate device memory for d_c");

  // Transfer data from host to device
  err = hipMemcpy(d_a, h_a, N * sizeof(int),
                   hipMemcpyKind::hipMemcpyHostToDevice);
  CheckCudaError(err, "Failed to copy h_a to d_a");

  err = hipMemcpy(d_b, h_b, N * sizeof(int),
                   hipMemcpyKind::hipMemcpyHostToDevice);
  CheckCudaError(err, "Failed to copy h_b to d_b");

  // Define the block size and grid size
  int blockSize{256};
  int gridSize{static_cast<int>((N + blockSize - 1) / blockSize)};

  // Calling the kernel
  // VectorAddKernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, N);
  VectorAddKernel<<<gridSize, blockSize, 0, stream>>>(d_a, d_b, d_c, N);

  // Check for kernel errors
  err = hipGetLastError();
  CheckCudaError(err, "Kernel execution failed");

  // Transfer data from device to host
  err = hipMemcpy(h_c, d_c, N * sizeof(int),
                   hipMemcpyKind::hipMemcpyDeviceToHost);
  CheckCudaError(err, "Failed to copy d_c to h_c");

  // Free device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}

}  // namespace cuda_lab::timing