#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <cstdlib>
#include <iostream>

#include "matrix.h"
#include "matrix_multiply.h"

namespace cuda_lab::matrix_multiply {

__device__ float GetElement(Matrix const m, std::size_t const row,
                            std::size_t const col) {
  // return m.elements[row * m.stride + col];
  return m.elements[row * m.stride + col];
}

__device__ float SetElement(Matrix const m, std::size_t const row,
                            std::size_t const col, float value) {
  m.elements[row * m.stride + col] = value;
}

__device__ Matrix GetSubMatrix(Matrix const m, std::size_t const row,
                               std::size_t const col) {
  Matrix a_sub;
  a_sub.height = kBlockSize;
  a_sub.width = kBlockSize;
  a_sub.stride = m.stride;
  a_sub.elements = &m.elements[m.stride * kBlockSize * row + kBlockSize * col];

  return a_sub;
}
// __global__ void SharedABMultiply(Matrix const a, Matrix const b, Matrix c) {
//   auto block_row{blockIdx.x};
//   auto block_col{blockIdx.y};

//   // Each thread block computes one sub-matrix of c
//   auto c_sub{GetSubMatrix(c, block_row, block_col)};

//   // Each thread computes one element of c_sub by accumulating results into
//   // c_value
//   auto c_value{0.0f};

//   // Thread row and within the c_sub
//   auto row{threadIdx.y};
//   auto col{threadIdx.x};

//   for (std::size_t m{0}; m < a.width / BLOCK_SIZE; ++m) {
//     auto a_sub{GetSubMatrix(a, block_row, m)};
//     auto b_sub{GetSubMatrix(b, m, block_col)};

//     __shared__ float a_tile[BLOCK_SIZE]
//                            [BLOCK_SIZE];  // Shared memory for a sub-matrix
//     __shared__ float b_tile[BLOCK_SIZE]
//                            [BLOCK_SIZE];  // Shared memory for b sub-matrix

//     a_tile[row][col] = GetElement(a_sub, row, col);
//     b_tile[row][col] = GetElement(b_sub, row, col);

//     __syncthreads();  // Synchronize to make sure the tile is loaded

//     for (std::size_t e{0}; e < BLOCK_SIZE; ++e) {
//       c_value += a_tile[row][e] * b_tile[e][col];
//     }

//     __syncthreads();  // Synchronize to make sure the tile is loaded
//   }

//   SetElement(c_sub, row, col, c_value);  // Write the result to c
// }

__global__ void CoalescedMultiply(Matrix const a, Matrix const b, Matrix c) {
  // Share memory on chip
  __shared__ float a_tile[kBlockSize][kBlockSize];

  auto row{blockIdx.y * blockDim.y + threadIdx.y};
  auto col{blockIdx.x * blockDim.x + threadIdx.x};

  if (row >= a.height || col >= b.width) {
    return;
  }

  // Load a_tile from global memory to shared memory
  a_tile[threadIdx.y][threadIdx.x] = a.elements[row * a.width + threadIdx.x];

  // Synchronize to make sure the tile is loaded
  __syncthreads();

  auto sum{0.0f};

  // Perform the computation
  for (std::size_t i{0}; i < kBlockSize; ++i) {
    sum += a_tile[threadIdx.y][i] * b.elements[i * b.width + col];
  }

  c.elements[row * b.width + col] = sum;
}

__global__ void SimpleMultiply(Matrix const a, Matrix const b, Matrix c) {
  auto row{blockIdx.y * blockDim.y + threadIdx.y};
  auto col{blockIdx.x * blockDim.x + threadIdx.x};

  if (row >= a.height || col >= b.width) {
    return;
  }

  // Output sample kernel info
  if (row == a.height / 2 && col == b.width / 2) {
    printf("Sample kernel info:\n");
    printf(
        "Block (%d, %d), Thread (%d, %d)\nRow = blockIdx.y * blockDim.y + "
        "threadIdx.y = %d * %d + %d = %d\nCol = blockIdx.x * blockDim.x + "
        "threadIdx.x = %d * %d + %d = %d\n\n",
        blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, blockIdx.y,
        blockDim.y, threadIdx.y, row, blockIdx.x, blockIdx.y, threadIdx.x, col);
  }

  auto sum{0.0f};
  for (std::size_t i{0}; i < a.width; ++i) {
    sum += GetElement(a, row, i) * GetElement(b, i, col);
  }
  SetElement(c, row, col, sum);
}

void MatrixMultiply(Matrix const& h_a, Matrix const& h_b, Matrix& h_c,
                    MatrixMultiplyType const type) {
  Matrix d_a;
  d_a.width = d_a.stride = h_a.width;
  d_a.height = h_a.height;
  std::size_t size{h_a.width * h_a.height * sizeof(float)};
  hipError_t err{hipMalloc(&d_a.elements, size)};
  hipMemcpy(d_a.elements, h_a.elements, size, hipMemcpyHostToDevice);

  Matrix d_b;
  d_b.width = d_b.stride = h_b.width;
  d_b.height = h_b.height;
  size = h_b.width * h_b.height * sizeof(float);
  err = hipMalloc(&d_b.elements, size);
  hipMemcpy(d_b.elements, h_b.elements, size, hipMemcpyHostToDevice);

  Matrix d_c;
  d_c.width = d_c.stride = h_c.width;
  d_c.height = h_c.height;
  size = h_c.width * h_c.height * sizeof(float);
  err = hipMalloc(&d_c.elements, size);

  dim3 block_size(kBlockSize, kBlockSize);
  dim3 grid_size(DivUp(h_b.width, block_size.x),
                 DivUp(h_a.height, block_size.y));

  // Print kernel info
  std::cout << "Kernel info: " << std::endl;
  std::cout << "Data size: " << "row: " << h_a.height << " x col: " << h_b.width
            << std::endl;
  std::cout << "Element number: " << h_a.height * h_b.width << std::endl;
  std::cout << "Block size: " << block_size.x << " x " << block_size.y
            << std::endl;
  std::cout << "Grid size: " << grid_size.x << " x " << grid_size.y << std::endl
            << std::endl;

  // Create CUDA events
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record the start event
  hipEventRecord(start, 0);

  // Calling the kernel
  switch (type) {
    case MatrixMultiplyType::kSimple: {
      SimpleMultiply<<<grid_size, block_size>>>(d_a, d_b, d_c);
      break;
    }
    case MatrixMultiplyType::kCoalesced: {
      CoalescedMultiply<<<grid_size, block_size>>>(d_a, d_b, d_c);
      break;
    }
    case MatrixMultiplyType::kSharedAB: {
      // SharedABMultiply<<<grid_size, block_size>>>(d_a, d_b, d_c);
      break;
    }
    default: {
      break;
    }
  }

  // Record the stop event
  hipEventRecord(stop, 0);

  // Wait for the stop event to complete
  hipEventSynchronize(stop);

  switch (type) {
    case MatrixMultiplyType::kSimple:
      std::cout << "SimpleMultiply executed." << std::endl;
      break;

    case MatrixMultiplyType::kCoalesced:
      std::cout << "CoalescedMultiply executed." << std::endl;
      break;

    default:
      break;
  }

  // Calculate the elapsed time
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, start, stop);
  std::cout << "Kernel execution time: " << elapsed_time << " ms" << std::endl;

  // Calculate bandwidth
  // A and B matrices
  float data_size_gb{2.0f * h_a.height * h_a.stride * sizeof(float) /
                     (1024.0f * 1024.0f * 1024.0f)};
  data_size_gb += h_a.height * h_a.stride * sizeof(float) /
                  (1024.0f * 1024.0f * 1024.0f);             // C matrix
  float bandwidth{data_size_gb / (elapsed_time / 1000.0f)};  // Convert ms to s
  std::cout << "Kernel bandwidth: " << bandwidth << " GB/s" << std::endl
            << std::endl;

  // Destroy CUDA events
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipStreamSynchronize(0);  // Wait for the kernel to finish
  // Check for kernel errors
  err = hipGetLastError();
  CheckCudaError(err, "Kernel execution failed");

  // Transfer data from device to host
  err = hipMemcpy(h_c.elements, d_c.elements, size, hipMemcpyDeviceToHost);
  CheckCudaError(err, "Failed to copy d_c to h_c");

  // // Free device memory
  hipFree(d_a.elements);
  hipFree(d_b.elements);
  hipFree(d_c.elements);
}

}  // namespace cuda_lab::matrix_multiply